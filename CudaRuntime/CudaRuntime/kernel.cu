#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

using namespace std;

hipError_t maxWithCuda(int* max_of_mins, int* matr, unsigned int size);


__global__ void minsRows(int* max_of_mins, const int *matrix, unsigned int matrix_size)
{
    //printf("%i %i\n", blockIdx.x, blockDim.x);
    int thread_number = threadIdx.x;
    int block_number = blockIdx.x;
    int block_size = blockDim.x;

    int raw_number = (block_number * block_size + thread_number);
    int min_in_column = matrix[raw_number * matrix_size];
    for (int i = 1; i < matrix_size; i++) 
        if (matrix[raw_number * matrix_size + i] < min_in_column)
            min_in_column = matrix[raw_number * matrix_size + i];
    
    if (block_number == 0 && thread_number == 0)
        *max_of_mins = min_in_column;
    else
        if (*max_of_mins < min_in_column)
            *max_of_mins = min_in_column;
}

int main()
{
    // Размер матрицы кратный 1024
    const int matrSize = 20480, outputSize = 10;

    printf("Matrix size %d\n", matrSize);

    int *matrix = new int[matrSize * matrSize];
    int *rows_mins = new int[matrSize];
    int max_of_mins = 0;
    for (int i = 0; i < matrSize; i++) 
    {
        rows_mins[i] = 0;
        for (int j = 0; j < matrSize; j++) 
        {
            matrix[i * matrSize + j] = i + j;
            if (i < outputSize && j < outputSize)
                printf("%i ", matrix[i * matrSize + j]);
        }
        if (i < outputSize)
            printf("\n");
    }
    printf("\n");
    


    // Add in parallel.
    hipError_t cudaStatus = maxWithCuda(&max_of_mins, matrix, matrSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("Max of mins %i\n", max_of_mins);


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA
hipError_t maxWithCuda(int* max_of_mins, int *matr, unsigned int size)
{
    int *dev_matr = 0;
    int *dev_max_of_mins = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers  
    cudaStatus = hipMalloc((void**)&dev_matr, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!1");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_max_of_mins, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! with max");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_matr, matr, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }




    float elapsed = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    // Launch a kernel on the GPU with one thread for each column.
    minsRows <<<size / 1024, 1024>>>(dev_max_of_mins, dev_matr, size);


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("The elapsed time %.2f ms\n", elapsed);



    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(max_of_mins, dev_max_of_mins, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! with max");
        goto Error;
    }



    Error:
    hipFree(dev_max_of_mins);
    hipFree(dev_matr);
    
    return cudaStatus;
}
