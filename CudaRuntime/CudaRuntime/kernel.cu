﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

using namespace std;

hipError_t maxWithCuda(int* columns_mins, int* matr, unsigned int size);


__global__ void minsRows(int * rows_mins, const int *matrix)
{
    int thread_number = threadIdx.x;
    int matrix_size = blockDim.x;

    int min_in_column = matrix[thread_number * matrix_size];
    for (int i = 1; i < matrix_size; i++)
        if (matrix[thread_number * matrix_size + i] < min_in_column)
            min_in_column = matrix[thread_number * matrix_size + i];
   
    rows_mins[thread_number] = min_in_column;
}

int main()
{
    const int matrSize = 10;

    int *matrix = new int[matrSize * matrSize];
    int *rows_mins = new int[matrSize];
    for (int i = 0; i < matrSize; i++) 
    {
        rows_mins[i] = 0;
        for (int j = 0; j < matrSize; j++) 
        {
            matrix[i * matrSize + j] = i + j;
            printf("%i ", matrix[i * matrSize + j]);
        }
        printf("\n");
    }
    printf("\n");



    // Add vectors in parallel.
    hipError_t cudaStatus = maxWithCuda(rows_mins, matrix, matrSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    //Print output
    printf("Mins of rows: ");
    int max_matrix = rows_mins[0];
    for (int i = 1; i < matrSize; i++) 
    {
        printf("%i ", rows_mins[i]);
        if (rows_mins[i] > max_matrix)
            max_matrix = rows_mins[i];
    }
    printf("\n");

    printf("Max of mins %i\n", max_matrix);




    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA
hipError_t maxWithCuda(int *rows_mins, int *matr, unsigned int size)
{
    int* dev_rows_mins = 0;
    int *dev_matr = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers  
    cudaStatus = hipMalloc((void**)&dev_matr, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rows_mins, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_matr, matr, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_rows_mins, rows_mins, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    // Launch a kernel on the GPU with one thread for each column.
    minsRows <<<1, size>>>(dev_rows_mins, dev_matr);



    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }


    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(rows_mins, dev_rows_mins, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    Error:
    hipFree(dev_rows_mins);
    hipFree(dev_matr);
    
    return cudaStatus;
}
